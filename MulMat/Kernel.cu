#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "hip/hip_runtime.h"
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/video/tracking.hpp>
#include "stdafx.h"
//#include "nppdefs.h"
//#include <npp.h>
#include <chrono>  // for high_resolution_clock

typedef unsigned char uchar;
typedef unsigned int uint;

#define BLOCK_SIZE 32
#define CV_64FC1 double
#define CV_32F float
#define CV_8U uchar

int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__
int absGrad(int grad) {
	if (grad < 0) {
		return -1 * grad;
	}
	else {
		return grad;
	}
}

__global__ void Kernel_Sobel(uchar* img, uchar* imgout, int ImgWidth, int imgHeigh) // , int* maskX, int* maskY
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	
	int Index = (ImgNumLigne * ImgWidth) + (ImgNumColonne); // will be 3x greater

	//int nani = (ImgNumLigne * (ImgWidth / 3)) + ImgNumColonne;

	if ((ImgNumColonne < ImgWidth -2 ) && (ImgNumLigne < imgHeigh -2 )) //width / 3
	{
		//imgout[Index] = 50;

			//int y = ImgNumLigne; // change imgnumligne pour y
			//int x = ImgNumColonne;
			//int i = Index;
			////imgout ->>> int 
			int i = Index;
			int gradX = img[i] * -3 + img[i + 1] * 0 + img[i + 2] * 3;
			i = ((ImgNumLigne + 1) * ImgWidth) + (ImgNumColonne);
			gradX += img[i] * -10 + img[i + 1] * 0 + img[i + 2] * 10;
			i = ((ImgNumLigne + 2) * ImgWidth) + (ImgNumColonne);
			gradX += img[i] * -3 + img[i + 1] * 0 + img[i + 2] * 3;
			
			i = Index;
			int gradY = img[i] * -3 + img[i + 1] * -10 + img[i + 2] * -3;
			i = ((ImgNumLigne + 1) * ImgWidth) + (ImgNumColonne);
			gradY += img[i] * 0 + img[i + 1] * 0 + img[i + 2] * 0;
			i = ((ImgNumLigne + 2) * ImgWidth) + (ImgNumColonne);
			gradY += img[i] * 3 + img[i + 1] * 10 + img[i + 2] * 3;


			int grad = absGrad(gradX) + absGrad(gradY);
			int norm = grad * 0.0625;

			imgout[Index] = norm;
			////	Gradient X ne pas calculer * 0
			//int gradX = img[i] * -1 + img[i + 1] * 0 + img[i + 2] * 1;
			//i = ((ImgNumLigne + 1) * ImgWidth) + (ImgNumColonne);
			//gradX += img[i] * -2 + img[i + 1] * 0 + img[i + 2] * 2;
			//i = ((ImgNumLigne + 2) * ImgWidth) + (ImgNumColonne);
			//gradX += img[i] * -1 + img[i + 1] * 0 + img[i + 2] * 1;

			//i = (ImgNumLigne * ImgWidth) + (ImgNumColonne * 3);

			////	Gradient Y
			//int gradY = img[i] * -1 + img[i + 1] * -2 + img[i + 2] * -1;
			//i = ((ImgNumLigne + 1) * ImgWidth) + (ImgNumColonne);
			//gradY += img[i] * 0 + img[i + 1] * 0 + img[i + 2] * 0;
			//i = ((ImgNumLigne + 2) * ImgWidth) + (ImgNumColonne);
			//gradY += img[i] * 1 + img[i + 1] * 2 + img[i + 2] * 1;

			////	Gradient 
			//int gradient = abs(gradX) + abs(gradY);
			//int norm = gradient * 0.125;

			//imgout[i] = norm;
	}

	return;
}

extern "C" bool GPGPU_Sobel(cv::Mat* imgTresh, cv::Mat* Grayscale)
{

	//	1. Initialize data
	hipError_t cudaStatus;
	uchar* gDevImage;
	uchar* gDevImageOut;

	uint imageSize = imgTresh->rows * imgTresh->step1(); // will be x 3 greater 
	uint gradientSize = imgTresh->rows * imgTresh->cols * sizeof(uchar);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(iDivUp(imgTresh->cols, BLOCK_SIZE), iDivUp(imgTresh->rows, BLOCK_SIZE));

	//	2. Allocation data
	cudaStatus = hipMalloc(&gDevImage, imageSize);
	cudaStatus = hipMalloc(&gDevImageOut, gradientSize);

	//	3. Copy data on GPU
	cudaStatus = hipMemcpy(gDevImage, imgTresh->data, imageSize, hipMemcpyHostToDevice);

	// Record start time
	auto start = std::chrono::high_resolution_clock::now();
	
	//	4. Launch kernel
	Kernel_Sobel<<<dimGrid, dimBlock>>>(gDevImage, gDevImageOut, imgTresh->step1(), imgTresh->rows);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	// Record end time
	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = finish - start;
	std::cout << "Elapsed time GPGPU: " << elapsed.count() << " s\n";
	
	//	5. Copy data on CPU
	cudaStatus = hipMemcpy(Grayscale->data, gDevImageOut, gradientSize, hipMemcpyDeviceToHost);

	//	6. Free GPU memory
Error:
	hipFree(gDevImage);
	hipFree(gDevImageOut);

	return cudaStatus;
}
