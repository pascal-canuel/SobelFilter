#include "hip/hip_runtime.h" 

#include <stdio.h>

typedef unsigned char uchar;

//	If you want to have the .cu intellisense:
//	Adding '.cu' under c++ extension in visual studio settings would enable syntax highlighting for c++ 
//	keywords only. EDIT: It in Tools -> Options -> Text Editor -> File Extension type in cu and select 
//	Microsoft Visual C++ as the editor and click add

/************************************************************************
// KERNEL qui permet de faire une multiplication scalaire d'une matrice
// d'entier. Chaque thread s'occupe d'un r�sultat
/***********************************************************************/
__global__
static void Kernel_ScalaireMulMat_Int(uchar *MatI, int K, uchar *MatO)
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int ImageWidth = blockDim.x * gridDim.x;
	int Index = ImgNumLigne * ImageWidth + ImgNumColonne;

	MatO[Index] = MatI[Index] + 50;
}

/************************************************************************
// Fonction de lancement du kernel qui permet de faire une multiplication
// scalaire d'une matrice d'entier.
/***********************************************************************/
extern "C" hipError_t Launcher_ScalaireMulMat_Int(uchar *pMatI, int K, uchar *pMatO, dim3 DimMat)
{
	//	1. Initialize data
	//	Choose which GPU to run on, change this on a multi-GPU system.    
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	int BLOCK_SIZE = 16;
	uchar *gMatI, *gMatO;

	//	Grid of BLOCK_SIZE * BLOCK_SIZE blocks
	dim3 dimGrid(BLOCK_SIZE, BLOCK_SIZE);
	//	Block of BLOCK_SIZE * BLOCK_SIZE threads
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	size_t memSize = DimMat.x * DimMat.y * sizeof(uchar);

	//	2. Allocate memory for the data on the GPU
	cudaStatus = hipMalloc(&gMatI, memSize);
	cudaStatus = hipMalloc(&gMatO, memSize);

	//	3. Copy the data on the GPU
	cudaStatus = hipMemcpy(gMatI, pMatI, memSize, hipMemcpyHostToDevice);

	//	4. Launch kernel
	Kernel_ScalaireMulMat_Int <<<dimGrid, dimBlock >>>(gMatI, K, gMatO);
	cudaStatus = hipDeviceSynchronize();	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel failed!");
		return cudaStatus;
	}

	//	5. Copy the data back on the CPU
	hipMemcpy(pMatO, gMatO, memSize, hipMemcpyDeviceToHost);

	//	6. Free the memory of the GPU
	hipFree(gMatI);
	hipFree(gMatO);

	return cudaStatus;
}
