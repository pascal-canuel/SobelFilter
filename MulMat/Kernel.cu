#include "hip/hip_runtime.h" 

#include <stdio.h>

//	If you want to have the .cu intellisense:
//	Adding '.cu' under c++ extension in visual studio settings would enable syntax highlighting for c++ 
//	keywords only. EDIT: It in Tools -> Options -> Text Editor -> File Extension type in cu and select 
//	Microsoft Visual C++ as the editor and click add

extern "C" hipError_t ConvolutionCuda(int *ImageIn,  int *ImageOut, int *Kernel, size_t ImageSize) 
{ 
	int *MatIn, *MatOut;
   // Choose which GPU to run on, change this on a multi-GPU system.    
   hipError_t cudaStatus = hipSetDevice(0);    
   if (cudaStatus != hipSuccess) {   
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
		goto Error;  
   } 
   
   Error:
	return cudaStatus;
}

