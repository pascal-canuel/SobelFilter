#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#include ""
#include <stdio.h>

typedef unsigned char uchar

//	If you want to have the .cu intellisense:
//	Adding '.cu' under c++ extension in visual studio settings would enable syntax highlighting for c++ 
//	keywords only. EDIT: It in Tools -> Options -> Text Editor -> File Extension type in cu and select 
//	Microsoft Visual C++ as the editor and click add

extern "C" hipError_t ConvolutionCuda(int *ImageIn,  int *ImageOut, int *Kernel, size_t ImageSize) 
{ 
   // Choose which GPU to run on, change this on a multi-GPU system.    
   hipError_t cudaStatus = hipSetDevice(0);    


   if (cudaStatus != hipSuccess) {   
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
		goto Error;  
   } 
   
   Error:
	return cudaStatus;
}

/************************************************************************
// KERNEL qui permet de faire une multiplication scalaire d'une matrice
// d'entier. Chaque thread s'occupe d'un r�sultat
/***********************************************************************/
__global__ 
static void Kernel_ScalaireMulMat_Int(int *MatA, int K, int *MatR, dim3 DimMat)
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int ImageWidth = blockDim.x * gridDim.x;
	int Index = ImgNumLigne * ImageWidth + ImgNumColonne;

	MatR[Index] = MatA[Index] + K;
	return;
}

/************************************************************************
// Fonction de lancement du kernel qui permet de faire une multiplication
// scalaire d'une matrice d'entier.
/***********************************************************************/
extern "C" hipError_t Launcher_ScalaireMulMat_Int(int *pMatA, int K, int *pMatR, dim3 DimMat)
{
	int BLOCK_SIZE = 32; //	Should be defined
	int *MatA, *MatR;
	dim3 dimBlock(DimMat.x, DimMat.y);
	//dim3 dimGrid(iDivUp(DimMat.x, BLOCK_SIZE), iDivUp(DimMat.y, BLOCK_SIZE)); 	
	dim3 dimGrid(BLOCK_SIZE, BLOCK_SIZE);
	hipError_t cudaStatus; 
	// Partir un timer pour calculer le temps d'ex�cution 
	unsigned int timer = 0; float TempsExecution;  
	// Allouer l'espace memoire des 2 matrices sur la carte GPU 
	size_t memSize = DimMat.x * DimMat.y * sizeof(int); 
	hipMalloc( (void **) &MatA, memSize ); 
	hipMalloc( (void **) &MatR, memSize ); 
	// Copier de la matrice A dans la memoire du GPU 
	hipMemcpy( MatA, pMatA, memSize, hipMemcpyHostToDevice ); 
	// Partir le kernel. ************* Sur une seul ligne  ********* 
	Kernel_ScalaireMulMat_Int<<<dimGrid,dimBlock>>>((int*)MatA,(int)K,(int*)MatR, DimMat);
	//CUT_CHECK_ERROR("Kernel execution failed\n"); 
	// Attendre la fin du kernel  
	cudaStatus = hipDeviceSynchronize();  
	if (cudaStatus != hipSuccess) {   
		fprintf(stderr, "Kernel  BackGroundSoustractionHSV failed!");
		return cudaStatus;  
	}
	// Transfert de la matrice r�sultat 
	//CUDA_SAFE_CALL( hipMemcpy(pMatR, MatR, memSize, hipMemcpyDeviceToHost));
	hipMemcpy(pMatR, MatR, memSize, hipMemcpyDeviceToHost);
	// Lib�rer la m�moire du 
	//GPU CUDA_SAFE_CALL( hipFree(MatA)); 
	hipFree(MatA);
	return cudaStatus;
}